
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

static int BLOCK_SIZE = 16;

static void fill_random(float *A, const int64_t n, const int64_t m) {

  int64_t i, j;
  for (i = 0; i < n; ++i) {
    for (j = 0; j < m; ++j) {
      A[i * m + j] = (float)rand() / (float)RAND_MAX;
    }
  }
}

__global__ void gemm(float *A, float *B, float *C, int64_t A_rows,
                     int64_t A_cols, int64_t B_cols) {

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  float sum = 0.0f;
  int i;

  if (col < B_cols && row < A_rows) {
    for (i = 0; i < A_cols; ++i) {
      sum += A[row * A_cols + i] * B[i * B_cols + col];
    }
    C[row * B_cols + col] = sum;
  }
}

static struct timespec print_dtime(struct timespec start, const char *process) {
  struct timespec end;
  clock_gettime(CLOCK_MONOTONIC_RAW, &end);
  const float dtime = ((float)((end.tv_sec - start.tv_sec) * 1000000 +
                               (end.tv_nsec - start.tv_nsec) / 1000)) /
                      1E6;

  printf("Time to %s = %f s\n", process, dtime);
  return end;
}

static void print_matrix(float *A, const int64_t A_rows, const int64_t A_cols) {

  int64_t i, j;
  printf("[");
  for (i = 0; i < A_rows; ++i) {
    for (j = 0; j < A_cols; ++j) {
      printf("%f, ", A[i * A_cols + j]);
    }
  }
  printf("]\n");
}

int main(int argc, char *argv[]) {

  // Assign seed from current time integer
  time_t t;
  srand((unsigned int)time(&t));

  int64_t A_rows, A_cols, B_rows, B_cols;
  int32_t steps = 1;

  if (argc == 5) {
    A_rows = atoll(argv[1]);
    A_cols = atoll(argv[2]);
    B_rows = atoll(argv[2]);
    B_cols = atoll(argv[3]);
    steps = atoll(argv[4]);
  } else if (argc == 4) {
    A_rows = atoll(argv[1]);
    A_cols = atoll(argv[2]);
    B_rows = atoll(argv[2]);
    B_cols = atoll(argv[3]);
  } else {
    printf("Usage: \n"
           "- 3 arguments: matrix A rows, matrix A cols and matrix B cols\n"
           "- 4 arguments: matrix A rows, matrix A cols and matrix B cols, "
           "steps\n");
    return 1;
  }

  printf("[ %ld %ld %ld ]\n", A_rows, A_cols, B_cols);

  struct timespec start, tmp;
  clock_gettime(CLOCK_MONOTONIC_RAW, &start);

  float *A_h = (float *)malloc((size_t)A_rows * (size_t)A_cols * sizeof(float));
  tmp = print_dtime(start, "allocate A");

  float *B_h = (float *)malloc((size_t)B_rows * (size_t)B_cols * sizeof(float));
  tmp = print_dtime(tmp, "allocate B");

  float *C_h = (float *)malloc((size_t)A_rows * (size_t)B_cols * sizeof(float));
  tmp = print_dtime(tmp, "allocate C");

  // value-init to zero
  fill_random(A_h, A_rows, A_cols);
  tmp = print_dtime(tmp, "fill A");

  fill_random(B_h, B_rows, B_cols);
  tmp = print_dtime(tmp, "fill B");

  // Allocate memory space on the device
  float *A_d, *B_d, *C_d;
  if (hipMalloc((void **)&A_d, sizeof(float) * A_rows * A_cols)) {
    printf("A_d allocation failure\n");
    exit(1); // leaky exit
  }
  tmp = print_dtime(tmp, "allocate A_d");

  if (hipMalloc((void **)&B_d, sizeof(float) * B_rows * B_cols)) {
    printf("B_d allocation failure\n");
    exit(1); // leaky exit
  }
  tmp = print_dtime(tmp, "allocate B_d");

  if (hipMalloc((void **)&C_d, sizeof(float) * A_rows * B_cols)) {
    printf("C_d allocation failure\n");
    exit(1); // leaky exit
  }
  tmp = print_dtime(tmp, "allocate C_d");

  hipMemcpy(A_d, A_h, sizeof(float) * A_rows * A_cols, hipMemcpyHostToDevice);
  tmp = print_dtime(tmp, "copy A");

  hipMemcpy(B_d, B_h, sizeof(float) * B_rows * B_cols, hipMemcpyHostToDevice);
  tmp = print_dtime(tmp, "copy B");

  unsigned int grid_rows = (A_rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int grid_cols = (B_cols + BLOCK_SIZE - 1) / BLOCK_SIZE;

  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  int32_t i;
  for (i = 0; i < steps; ++i) {
    gemm<<<dimGrid, dimBlock>>>(A_d, B_d, C_d, A_rows, A_cols, B_cols);
    hipDeviceSynchronize();
    tmp = print_dtime(tmp, "simple gemm");
  }

  hipMemcpy(C_h, C_d, sizeof(float) * A_rows * B_cols, hipMemcpyDeviceToHost);
  tmp = print_dtime(tmp, "copy C");

  // print_matrix(A_h, A_rows, A_cols);
  // print_matrix(B_h, B_rows, B_cols);
  // print_matrix(C_h, A_rows, B_cols);

  print_dtime(start, "total time");

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);

  free(A_h);
  free(B_h);
  free(C_h);

  return 0;
}
