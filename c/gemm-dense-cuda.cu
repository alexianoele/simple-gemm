#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

static int BLOCK_SIZE = 16;

static void fill_random(float *A, const int64_t n, const int64_t m) {

  int64_t i, j;
  for (i = 0; i < n; ++i) {
    for (j = 0; j < m; ++j) {
      A[i * m + j] = (float)rand() / (float)RAND_MAX;
    }
  }
}

__global__ 
void gemm(float *A, float *B, float *C, const int64_t A_rows,
                     const int64_t A_cols, const int64_t B_cols) {

    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += A[row * n + i] * B[i * k + col];
        }
        C[row * k + col] = sum;
    }

}

static struct timespec print_dtime(struct timespec start, const char *process) {
  struct timespec end;
  clock_gettime(CLOCK_MONOTONIC_RAW, &end);
  const float dtime = ((float)((end.tv_sec - start.tv_sec) * 1000000 +
                               (end.tv_nsec - start.tv_nsec) / 1000)) /
                      1E6;

  printf("Time to %s = %f s\n", process, dtime);
  return end;
}

static void print_matrix(float *A, const int64_t A_rows, const int64_t A_cols) {

  int64_t i, j;
  printf("[");
  for (i = 0; i < A_rows; ++i) {
    for (j = 0; j < A_cols; ++j) {
      printf("%f, ", A[i * A_cols + j]);
    }
  }
  printf("]\n");
}

int main(int argc, char *argv[]) {

  // Assign seed from current time integer
  time_t t;
  srand((unsigned int)time(&t));

  int64_t A_rows, A_cols, B_rows, B_cols;

  if (argc != 4) {
    printf(
        "Usage: 3 arguments: matrix A rows, matrix A cols and matrix B cols\n");
    return 1;
  } else {
    A_rows = atoll(argv[1]);
    A_cols = atoll(argv[2]);
    B_rows = atoll(argv[2]);
    B_cols = atoll(argv[3]);
  }

  printf("[ %ld %ld %ld ]\n", A_rows, A_cols, B_cols);

  struct timespec start, tmp;
  clock_gettime(CLOCK_MONOTONIC_RAW, &start);

  float* A_h, B_h, C_h;
  
  hipHostMalloc( ( void**) &A_h, sizeof(float) * A_rows * A_cols);
  tmp = print_dtime(start, "allocate A");

  hipHostMalloc( ( void**) &B_h, sizeof(float) * B_rows * B_cols);
  tmp = print_dtime(tmp, "allocate B");

  hipHostMalloc( ( void**) &C_h, sizeof(float) * A_rows * B_cols);
  tmp = print_dtime(tmp, "allocate C"); 
  
  // value-init to zero
  fill_random(A_h, A_rows, A_cols);
  tmp = print_dtime(tmp, "fill A");

  fill_random(B_h, B_rows, B_cols);
  tmp = print_dtime(tmp, "fill B");

  
  // Allocate memory space on the device
  double *A_d, *B_d, *C_d;
  hipMalloc((void **) &A_d, sizeof(float) * A_rows * A_cols );
  tmp = print_dtime(tmp, "allocate A_d");
  hipMalloc((void **) &B_d, sizeof(float) * B_rows * B_cols );
  tmp = print_dtime(tmp, "allocate B_d");
  hipMalloc((void **) &C_d, sizeof(float) * A_rows * B_cols );
  tmp = print_dtime(tmp, "allocate C_d");

  hipMemcpy(A_d, A_h, sizeof(float)*A_rows*A_cols, cudaMemcpuHostToDevice);
  tmp = print_dtime(tmp, "copy A");

  hipMemcpy(B_d, B_h, sizeof(float)*B_rows*B_cols, cudaMemcpuHostToDevice);
  tmp = print_dtime(tmp, "copy B");
 
  unsigned int grid_rows = (A_rows + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int grid_cols = (B_cols + BLOCK_SIZE - 1) / BLOCK_SIZE;
  dim3 dimGrid(grid_cols, grid_rows);
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

  gemm <<<dimGrid, dimBlock>>>( A_d, B_d, C_d, A_rows, A_cols, B_cols);
  tmp = print_dtime(tmp, "simple gemm");

  hipMemcpy(C_h, C_d, sizeof(float)*A_rows*B_cols, cudaMemcpuDeviceToHost);
  tmp = print_dtime(tmp, "copy C");

  print_dtime(start, "total time");

  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);

  free(A_h);
  free(B_h);
  free(C_h);

  return 0;
}
